#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdint.h>
#include <iostream>
#include <array>
#include <cstddef>

#include <glad/glad.h>
#include <GLFW/glfw3.h>

#include "ImGUI/imgui.h"
#include "ImGUI/imgui_impl_glfw.h"
#include "ImGUI/imgui_impl_opengl3.h"

#include "cuda_gl_interop.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include "Types.h"
#include "Shader.h"
#include "TripRandom.h"
#include "ComputeShader.h"

static const char *_cudaGetErrorEnum(hipError_t error)
{
	return hipGetErrorName(error);
}

template <typename T>
void check(T result, char const *const func, const char *const file,int const line)
{
	if (result) 
	{
		fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
			static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
		exit(EXIT_FAILURE);
	}
}
#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

void GLAPIENTRY
MessageCallback(GLenum source,
	GLenum type,
	GLuint id,
	GLenum severity,
	GLsizei length,
	const GLchar* message,
	const void* userParam)
{
	if (severity > GL_DEBUG_SEVERITY_LOW)
		fprintf(stderr, "GL CALLBACK: %s type = 0x%x, severity = 0x%x, message = %s\n",
		(type == GL_DEBUG_TYPE_ERROR ? "** GL ERROR **" : ""),
			type, severity, message);
}

template<class T>
constexpr const T& clamp(const T& v, const T& lo, const T& hi)
{
	assert(!(hi < lo));
	return (v < lo) ? lo : (hi < v) ? hi : v;
}

constexpr u32 SIZE = 1024;
constexpr u32 WIDTH = 840;
constexpr u32 HEIGHT = 600;

__device__ float generate(hiprandState* globalState, int ind)
{
	//int ind = threadIdx.x;
	hiprandState localState = globalState[ind%5];
	float RANDOM = hiprand_uniform(&localState);
	globalState[ind%5] = localState;
	return RANDOM;
}

__global__ void RandInit(hiprandState* state, int seed)
{
	hiprand_init(seed, threadIdx.x,  0, &state[threadIdx.x]);
}


__global__ void ParticleSim(f32 dt,
	float3* pos,
	float3* vel,
	float1* life,
	float3* col,
	hiprandState* state,
	float3 ParticleColor,
	bool bSpawn
)
{
	u32 i = threadIdx.x + blockIdx.x * blockDim.x;

	v3 inPos = { pos[i].x, pos[i].y, pos[i].z };
	v3 inCol = { col[i].x, col[i].y, col[i].z };
	v3 inVel = { vel[i].x, vel[i].y, vel[i].z };
	f32 inLife = { life[i].x - dt};
	
	if (bSpawn && inLife < 0.0f)
	{
		float r = generate(state, i) * 2.0 * 3.1415926;
		inPos = v3(cos(r)*10.1f, sin(r)*10.1f, 0.f);
		//inPos = v3(0.f, generate(state,i) * 6.0f - 3.0f, 0.f);
		//inPos = v3(generate(state, i));
		inLife = 6.0f * generate(state,i);
		//inCol = v3(generate(state, i), generate(state, i) * .5f, generate(state, i) * 0.3f) * 0.5f + 0.5f;
		inCol = v3(ParticleColor.x, ParticleColor.y, ParticleColor.z) * v3(generate(state, i)) * 0.75f + 0.25f;
		inVel = v3(generate(state, i)* 2.0f - 1.0f, generate(state, i), generate(state, i) * 2.0f - 1.0f);
	}
	inCol = inCol;// *glm::min(1.0f, inLife);
	inPos = inPos + (inVel * dt);
	//inVel += v3(cosf(inPos.x), 0., sinf(inPos.z));
	//inVel += v3(inPos.z * inPos.z * inPos.z, 0., inPos.x * inPos.x * inPos.x);

	f32 t = atan2(inPos.z, inPos.x);
	//f32 phi = atan2(glm::length(inPos) , inPos.y);
	inVel += v3(sinf(t), -inPos.y*.3f, -cosf(t))* dt * 10.f;
	inVel += -glm::normalize(inPos) * dt;
	//inVel += v3(0,0,-10) * dt;
	//inVel += v3(-2.f * inPos.x + 2.f * inPos.z , 0, -3.f * inPos.x - 3.f * inPos.z) * dt * 2.f;
	
	pos[i] = make_float3(inPos.x, inPos.y, inPos.z);
	col[i] = make_float3(inCol.x, inCol.y, inCol.z);
	vel[i] = make_float3(inVel.x, inVel.y - dt * .3f, inVel.z);
	life[i].x = inLife;
}

__global__ void InvertColor(float4* pbo)
{
	u32 i = threadIdx.x + blockIdx.x * blockDim.x;
	pbo[i] = make_float4(1.0f - pbo[i].x, 1.0f - pbo[i].y, 1.0f - pbo[i].z, 1.0f);
}

static bool s_Run = true;

unsigned int quadVAO = 0;
unsigned int quadVBO;
void renderQuad()
{
	if (quadVAO == 0)
	{
		float quadVertices[] = {
			// positions        // texture Coords
			-1.0f,  1.0f, 0.0f, 0.0f, 1.0f,
			-1.0f, -1.0f, 0.0f, 0.0f, 0.0f,
			 1.0f,  1.0f, 0.0f, 1.0f, 1.0f,
			 1.0f, -1.0f, 0.0f, 1.0f, 0.0f,
		};
		// setup plane VAO
		glGenVertexArrays(1, &quadVAO);
		glGenBuffers(1, &quadVBO);
		glBindVertexArray(quadVAO);
		glBindBuffer(GL_ARRAY_BUFFER, quadVBO);
		glBufferData(GL_ARRAY_BUFFER, sizeof(quadVertices), &quadVertices, GL_STATIC_DRAW);
		glEnableVertexAttribArray(0);
		glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)0);
		glEnableVertexAttribArray(1);
		glVertexAttribPointer(1, 2, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)(3 * sizeof(float)));
	}
	glBindVertexArray(quadVAO);
	glDrawArrays(GL_TRIANGLE_STRIP, 0, 4);
	glBindVertexArray(0);
}

enum class eProgramState : u32
{
	NORMAL, SLOW_MO, PAUSED, SIZE
};

int main()
{
	eProgramState ProgState = eProgramState::NORMAL;
	
	glfwInit();
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 4);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
	glfwWindowHint(GLFW_SAMPLES, 4);
	// glfw window creation
	// --------------------
	GLFWwindow* window = glfwCreateWindow(WIDTH, HEIGHT, "CudaGL", nullptr, nullptr);
	if (window == nullptr)
	{
		std::cerr << "Failed to create GLFW window.\n";
		glfwTerminate();
	}
	else
	{
		std::cerr << "GLFW Create Window Succesful!\n";
		glfwMakeContextCurrent(window);
	}

	GLFWmonitor* monitor = glfwGetPrimaryMonitor();
	const GLFWvidmode *mode = glfwGetVideoMode(monitor);
	if (mode)
	{
		i32 monitorX, monitorY;
		glfwGetMonitorPos(monitor, &monitorX, &monitorY);
		glfwSetWindowPos(window, monitorX + (mode->width - WIDTH) / 2, monitorY + (mode->height - HEIGHT) / 2);
	}

	// load opengl
	if (!gladLoadGLLoader(reinterpret_cast<GLADloadproc>(glfwGetProcAddress)))
	{
		std::cerr << "Failed to initialize GLAD\n";
		std::cin.get();
		exit(1);
	}

	glfwSetWindowCloseCallback(window, [](GLFWwindow* window) {s_Run = false; });

	struct WindowData
	{
		f32 Scroll = {};
	} winData;

	glfwSetWindowUserPointer(window, &winData);

	glfwSetScrollCallback(window, [](GLFWwindow* win, f64 x, f64 y)
	{
		WindowData& data = *static_cast<WindowData*>(glfwGetWindowUserPointer(win));
		data.Scroll = y;
	});

	IMGUI_CHECKVERSION();
	ImGui::CreateContext();
	ImGuiIO& io = ImGui::GetIO(); (void)io;
	io.ConfigFlags |= ImGuiConfigFlags_NavEnableKeyboard;       // Enable Keyboard Controls
	//io.ConfigFlags |= ImGuiConfigFlags_NavEnableGamepad;      // Enable Gamepad Controls
	io.ConfigFlags |= ImGuiConfigFlags_DockingEnable;           // Enable Docking
	io.ConfigFlags |= ImGuiConfigFlags_ViewportsEnable;         // Enable Multi-Viewport / Platform Windows

	//io.ConfigViewportsNoAutoMerge = true;
	//io.ConfigViewportsNoTaskBarIcon = true;

	io.DisplaySize = ImVec2(WIDTH, HEIGHT);

	// Setup Dear ImGui style
	ImGui::StyleColorsDark();
	//ImGui::StyleColorsClassic();

	// When viewports are enabled we tweak WindowRounding/WindowBg so platform windows can look identical to regular ones.
	ImGuiStyle& style = ImGui::GetStyle();
	if (io.ConfigFlags & ImGuiConfigFlags_ViewportsEnable)
	{
		style.WindowRounding = 0.0f;
		style.Colors[ImGuiCol_WindowBg].w = 1.0f;
	}

	// Setup Platform/Renderer bindings
	ImGui_ImplGlfw_InitForOpenGL(window, true);
	ImGui_ImplOpenGL3_Init("#version 130");


	
	// init cuda rand
	hiprandState* devStates;
	hipMalloc(&devStates, 5 * sizeof(hiprandState));
	RandInit<<<1,5>>>(devStates, TripRandom::Float() * 100);
	
	printf("OPENGL VERSION: %s\n", glGetString(GL_VERSION));

	glEnable(GL_DEPTH_TEST);
	glEnable(GL_DEBUG_OUTPUT);
	glDebugMessageCallback(MessageCallback, nullptr);
	glPointSize(3.0f);

//	glBlendFunc(GL_ONE, GL_ONE);

	// setup particle system 
	hipGraphicsResource *cuda_pos_vbo_res;
	hipGraphicsResource *cuda_col_vbo_res;
	hipGraphicsResource *cuda_vel_vbo_res;
	hipGraphicsResource *cuda_life_vbo_res;

	u32 VAO;
	u32 posVBO;
	u32 colVBO;
	u32 velVBO;
	u32 lifeVBO;
	
	//constexpr auto ParticleCount = 40'000'000;
	constexpr auto ParticleCount = 40'000'000;
	
	glGenVertexArrays(1, &VAO);
	glBindVertexArray(VAO);

	glGenBuffers(1, &posVBO);
	glBindBuffer(GL_ARRAY_BUFFER, posVBO);
	glBufferData(GL_ARRAY_BUFFER, ParticleCount * sizeof(v3), 0, GL_DYNAMIC_DRAW); // posVBO_Buffer.data(), GL_DYNAMIC_DRAW);
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 3 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(0);
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_pos_vbo_res, posVBO, cudaGraphicsMapFlagsNone));

	glGenBuffers(1, &colVBO);
	glBindBuffer(GL_ARRAY_BUFFER, colVBO);
	glBufferData(GL_ARRAY_BUFFER, ParticleCount * sizeof(v3), 0, GL_DYNAMIC_DRAW); 
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 3 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(1);
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_col_vbo_res, colVBO, cudaGraphicsMapFlagsNone));
	
	glBindVertexArray(0);

	glGenBuffers(1, &velVBO);
	glBindBuffer(GL_ARRAY_BUFFER, velVBO);
	glBufferData(GL_ARRAY_BUFFER, ParticleCount * sizeof(v3), 0, GL_DYNAMIC_DRAW);
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vel_vbo_res, velVBO, cudaGraphicsMapFlagsNone));

	glGenBuffers(1, &lifeVBO);
	glBindBuffer(GL_ARRAY_BUFFER, lifeVBO);
	glBufferData(GL_ARRAY_BUFFER, ParticleCount * sizeof(f32), 0, GL_DYNAMIC_DRAW);
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_life_vbo_res, lifeVBO, cudaGraphicsMapFlagsNone));

	glBindBuffer(GL_ARRAY_BUFFER, 0);

	// setup framebuffers
	u32 FBO;
	u32 FBOColor[2]; // 0 : Scene color, 1 : scene brightness
	hipGraphicsResource *cuda_fbo_res[2];

	glGenFramebuffers(1, &FBO);
	glBindFramebuffer(GL_FRAMEBUFFER, FBO);
	
	glGenTextures(2, FBOColor);

	for (unsigned int i = 0; i < 2; i++)
	{
		glBindTexture(GL_TEXTURE_2D, FBOColor[i]);

		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA16F, WIDTH, HEIGHT, 0, GL_RGBA, GL_FLOAT, NULL);

		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
		glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0 + i, GL_TEXTURE_2D, FBOColor[i], 0);
		//checkCudaErrors(hipGraphicsGLRegisterImage(&cuda_fbo_res[i], FBOColor[i], GL_TEXTURE_2D, cudaGraphicsMapFlagsNone));
	}
	glBindTexture(GL_TEXTURE_2D, 0);

	unsigned int attachments[2] = { GL_COLOR_ATTACHMENT0, GL_COLOR_ATTACHMENT1 };
	glDrawBuffers(2, attachments);
	//glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_2D, FBOColor, 0);

	u32 rbo;
	glGenRenderbuffers(1, &rbo);
	glBindRenderbuffer(GL_RENDERBUFFER, rbo);
	glRenderbufferStorage(GL_RENDERBUFFER, GL_DEPTH24_STENCIL8, WIDTH, HEIGHT);
	glBindRenderbuffer(GL_RENDERBUFFER, 0);
	glFramebufferRenderbuffer(GL_FRAMEBUFFER, GL_DEPTH_STENCIL_ATTACHMENT, GL_RENDERBUFFER, rbo);
	if (glCheckFramebufferStatus(GL_FRAMEBUFFER) != GL_FRAMEBUFFER_COMPLETE)
		std::cout << "ERROR::FRAMEBUFFER:: Framebuffer is not complete!" << std::endl;
	glBindFramebuffer(GL_FRAMEBUFFER, 0);

	// post processing framebuffers
	unsigned int pingpongFBO[2];
	unsigned int pingpongColorbuffers[2];
	glGenFramebuffers(2, pingpongFBO);
	glGenTextures(2, pingpongColorbuffers);
	for (unsigned int i = 0; i < 2; i++)
	{
		glBindFramebuffer(GL_FRAMEBUFFER, pingpongFBO[i]);
		glBindTexture(GL_TEXTURE_2D, pingpongColorbuffers[i]);
		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB16F, WIDTH, HEIGHT, 0, GL_RGB, GL_FLOAT, NULL);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE); // we clamp to the edge as the blur filter would otherwise sample repeated texture values!
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
		glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_2D, pingpongColorbuffers[i], 0);

		// also check if framebuffers are complete (no need for depth buffer)
		if (glCheckFramebufferStatus(GL_FRAMEBUFFER) != GL_FRAMEBUFFER_COMPLETE)
			std::cout << "Framebuffer not complete!" << std::endl;
	}
	glBindFramebuffer(GL_FRAMEBUFFER, 0);
	glBindTexture(GL_TEXTURE_2D, 0);

	static f32 sCurTime = glfwGetTime();
	static f32 sRunTime = 0.0f;
	static f32 sDT = 0.0f;

	// Camera
	m4 Proj = glm::perspective(45.0f, WIDTH / (f32)HEIGHT, 0.1f, 100.f);
	m4 View = glm::lookAt(v3(0, 0, 1), v3(0, 0, 0), v3(0, 1, 0));
	
	f32 camTheta = 90.f;
	f32 camPhi = 1.6f;
	f32 camRho = 10.0f;
	f32 maxCampRho = 50.f;
	v3 camPos = {};

	auto GetShpericalCoord = [](f32 t, f32 phi, f32 rho) -> v3
	{
		return {
			rho * cosf(t) * sinf(phi),
			rho * cosf(phi),
			rho * sinf(t) * sinf(phi)
		};
	};

	v2 MousePos = {};
	v2 LastMousePos = {};

	// Load Shaders
	Shader screen_shader("screen");
	screen_shader.Bind();
	screen_shader.setInt("uTexture0", 0);
	screen_shader.setInt("uTexture1", 1);
	screen_shader.setInt("uExp", 1.0f);
	f32 screen_shader_uExp = 1.0f;

	Shader blur_shader("blur");
	blur_shader.Bind();
	blur_shader.setInt("uTexture0", 0);
	blur_shader.setInt("horizontal", 0);
	
	Shader shader("shader", true);
	shader.Bind();
	shader.setMat4("uP", Proj);

	//ComputeShader ParticleSimShader("particleCreater");

	// ImGui Config

	float3 ParticleColor = make_float3(0.2f,0.2f,1.0f);
	
	// main loop
	while (s_Run)
	{
		sDT = glfwGetTime() - sCurTime;

		ImGui_ImplOpenGL3_NewFrame();
		ImGui_ImplGlfw_NewFrame();
		ImGui::NewFrame();

		{
			ImGui::Begin("Settings");
			ImGui::Text("fps: %.2f, Number of Particles: %d", ImGui::GetIO().Framerate, ParticleCount);
			ImGui::DragFloat("Exposure", &screen_shader_uExp, 0.1);
			ImGui::ColorEdit3("Particle Color", (float*)&ParticleColor);
			ImGui::End();
		}
		
		glBindFramebuffer(GL_FRAMEBUFFER, FBO);
		
		//glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
		glClearColor(0.00002f, 0.00003f, 0.0003f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		
		// Input
		f64 mx, my;
		glfwGetCursorPos(window, &mx, &my);
		LastMousePos = MousePos;
		MousePos = { mx,my };

		if (glfwGetMouseButton(window, 1))
		{
			camPhi += (LastMousePos.y - MousePos.y) * sDT * 0.5f;
			camTheta -= (LastMousePos.x - MousePos.x) * sDT * 0.5f;
		}

		camRho += -winData.Scroll;
		winData.Scroll = 0.0f;

		if (glfwGetKey(window, GLFW_KEY_SPACE))
			ProgState = (ProgState == eProgramState::NORMAL) ? eProgramState::PAUSED : eProgramState::NORMAL;

		if (glfwGetKey(window, GLFW_KEY_C))
			ProgState = eProgramState::SLOW_MO;

		if (glfwGetKey(window, GLFW_KEY_I))
			screen_shader_uExp += 0.5 * sDT;
		if (glfwGetKey(window, GLFW_KEY_K))
			screen_shader_uExp -= 0.5 * sDT;
		
		if (glfwGetKey(window, GLFW_KEY_ESCAPE))
			s_Run = false;
		
		if (glfwGetKey(window, GLFW_KEY_A))
			camTheta += sDT;
		if (glfwGetKey(window, GLFW_KEY_D))
			camTheta -= sDT;

		if (glfwGetKey(window, GLFW_KEY_W))
			camPhi += sDT;
		if (glfwGetKey(window, GLFW_KEY_S))
			camPhi -= sDT;

		if (glfwGetKey(window, GLFW_KEY_R))
			camRho -= sDT;
		if (glfwGetKey(window, GLFW_KEY_F))
			camRho += sDT;

		if (glfwGetKey(window, GLFW_KEY_Z))
			maxCampRho += sDT;
		if (glfwGetKey(window, GLFW_KEY_X))
			maxCampRho -= sDT;

		bool bSpawn = true;
		if (glfwGetKey(window, GLFW_KEY_N))
			bSpawn = false;

		camRho = clamp(camRho, 0.1f, maxCampRho);
		camPhi = clamp(camPhi, 0.01f, glm::pi<f32>());
		
		camPos = GetShpericalCoord(camTheta, camPhi, camRho);
		View = glm::lookAt(camPos, v3(0, 0, 0), v3(0, 1, 0));
		shader.setMat4("uV", View);

		switch (ProgState)
		{
		case eProgramState::SLOW_MO:
			sDT *= 0.5f;
			break;
		case eProgramState::PAUSED:
			sDT = 0.0f;
			break;
		}

		sRunTime += sDT;
		sCurTime = glfwGetTime();

#if 1
		float3* d_pos_ptr;
		float3* d_col_ptr;
		float3* d_vel_ptr;
		float1* d_life_ptr;

		std::size_t num_bytes;
		checkCudaErrors(hipGraphicsMapResources(1, &cuda_pos_vbo_res, 0));
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&d_pos_ptr,  &num_bytes, cuda_pos_vbo_res));

		checkCudaErrors(hipGraphicsMapResources(1, &cuda_col_vbo_res, 0));
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&d_col_ptr, &num_bytes, cuda_col_vbo_res));
		
		checkCudaErrors(hipGraphicsMapResources(1, &cuda_vel_vbo_res, 0));
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&d_vel_ptr,  &num_bytes, cuda_vel_vbo_res));
		
		checkCudaErrors(hipGraphicsMapResources(1, &cuda_life_vbo_res, 0));
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&d_life_ptr, &num_bytes, cuda_life_vbo_res));

		//printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);

		ParticleSim<<<ParticleCount/1000, 1000>>>(sDT, 
			d_pos_ptr, d_vel_ptr, d_life_ptr, d_col_ptr, devStates, ParticleColor, bSpawn);

		checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_life_vbo_res, 0));
		checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vel_vbo_res, 0));
		checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_col_vbo_res, 0));
		checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_pos_vbo_res, 0));
#endif

		//ParticleSimShader.Bind(ParticleCount, posVBO, velVBO, lifeVBO, sDT);
		
		glBindVertexArray(VAO);
		glDrawArrays(GL_POINTS, 0, ParticleCount);
		glBindVertexArray(0);

		glBindFramebuffer(GL_FRAMEBUFFER, 0);

		auto err = glGetError();
		blur_shader.Bind();
		blur_shader.setInt("uTexture0", 0);
		bool horizontal = true, first_iteration = true;
		unsigned int amount = 10;
		glActiveTexture(GL_TEXTURE0);
		for (unsigned int i = 0; i < amount; i++)
		{
			glBindFramebuffer(GL_FRAMEBUFFER, pingpongFBO[horizontal]);
			blur_shader.setInt("horizontal", horizontal);
			glBindTexture(GL_TEXTURE_2D, first_iteration ? FBOColor[1] : pingpongColorbuffers[!horizontal]);  // bind texture of other framebuffer (or scene if first iteration)
			renderQuad();
			horizontal = !horizontal;
			first_iteration = false;
		}
		err = glGetError();
		if(err)
		{
			printf("Error: %i\n", err);
		}
		
		glBindFramebuffer(GL_FRAMEBUFFER, 0);
		
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		screen_shader.Bind();
		screen_shader.setFloat("uExp", screen_shader_uExp);
		glActiveTexture(GL_TEXTURE0);
		glBindTexture(GL_TEXTURE_2D, FBOColor[0]);
		glActiveTexture(GL_TEXTURE1);
		glBindTexture(GL_TEXTURE_2D, pingpongColorbuffers[!horizontal]);
		
		renderQuad(); // display final results
		shader.Bind();

		ImGui::Render();
		ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());
		
		glfwSwapBuffers(window);
		glfwPollEvents();
	}

	checkCudaErrors(hipGraphicsUnregisterResource(cuda_life_vbo_res));
	checkCudaErrors(hipGraphicsUnregisterResource(cuda_pos_vbo_res));
	checkCudaErrors(hipGraphicsUnregisterResource(cuda_col_vbo_res));
	checkCudaErrors(hipGraphicsUnregisterResource(cuda_vel_vbo_res));

	glfwTerminate();
}